#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

//each pixel has 4 values (0 - 255)
//R - RED
//G - GREEN
//B - BLUE
//Transparency (Alpha)


__global__ void Blureffect(unsigned char * inputImage, unsigned char * outputImage, unsigned int *width, unsigned int *height){

int r;
int g;
int b;
int a;

int threadID = blockDim.x * blockIdx.x + threadIdx.x;

int pixel = threadID * 4;

unsigned int w = *width;
unsigned int h = *height;

r = inputImage[pixel];
g = inputImage[pixel+1];
b = inputImage[pixel+2];
a = inputImage[pixel+3];

if(threadID == 0) // Top Left corner
	{

		r = (inputImage[pixel] + inputImage[pixel+4] + inputImage[pixel + (4*w)] + inputImage[(pixel + (4 * w))+4]) / 4;
		g = (inputImage[pixel+1] + inputImage[(pixel+4)+1] + inputImage[(pixel + (4*w))+1] + inputImage[((pixel + (4 * w))+4)+1]) / 4;
		b = (inputImage[pixel+2] + inputImage[(pixel+4)+2] + inputImage[(pixel + (4*w))+2] + inputImage[((pixel + (4 * w))+4)+2]) / 4;

	} 
else if (threadID == w - 1) // Top Right
	{

		r = (inputImage[pixel] + inputImage[pixel - 4] + inputImage[pixel + (w * 4)] + inputImage[pixel + (w * 4) - 4]) / 4;
		g = (inputImage[pixel +1] + inputImage[pixel - 4 + 1] + inputImage[pixel + (w * 4) + 1] + inputImage[pixel + (w * 4) - 4 + 1]) / 4;
		b = (inputImage[pixel +2] + inputImage[pixel - 4 + 2] + inputImage[pixel + (w * 4) + 2] + inputImage[pixel + (w * 4) - 4 + 2]) / 4;
	
	}

else if (threadID == w * (h -1)) // Bottom left
	{
	
		r = (inputImage[pixel] + inputImage[pixel - (w * 4)] + inputImage[pixel - (w * 4) + 4] + inputImage[pixel + 4]) / 4;
		g = (inputImage[pixel+1] + inputImage[pixel - (w * 4)+1] + inputImage[pixel - (w * 4) + 4+1] + inputImage[pixel + 4+1]) / 4;
		b = (inputImage[pixel+2] + inputImage[pixel - (w * 4)+2] + inputImage[pixel - (w * 4) + 4+2] + inputImage[pixel + 4+2]) / 4;
	}
	
else if (threadID == (w * h)-1) // Bottom right
	{

		r = (inputImage[pixel] + inputImage[pixel - 4] + inputImage[pixel - (w * 4) - 4] + inputImage[pixel - (w * 4)]) / 4;
		g = (inputImage[pixel + 1] + inputImage[pixel - 4 + 1] + inputImage[pixel - (w * 4) - 4 + 1] + inputImage[pixel - (w * 4) + 1]) / 4;
		b = (inputImage[pixel + 2] + inputImage[pixel - 4 + 2] + inputImage[pixel - (w * 4) - 4 + 2] + inputImage[pixel - (w * 4) + 2]) / 4;

	}
	
else if (threadID > 0 && threadID < (w - 1)) // Top Row
	{
		
		r = (inputImage[pixel] + inputImage[pixel + 4] + inputImage[pixel + 4 + (w * 4)] + inputImage[pixel + (w * 4)] + inputImage[pixel - 4 + (w * 4)] + inputImage[pixel - 4]) / 6;
		g = (inputImage[pixel +1] + inputImage[pixel + 4 +1] + inputImage[pixel + 4 + (w * 4) +1] + inputImage[pixel + (w * 4) +1] + inputImage[pixel - 4 + (w * 4) +1] + inputImage[pixel - 4 +1]) / 6;
		b = (inputImage[pixel +2] + inputImage[pixel + 4 +2] + inputImage[pixel + 4 + (w * 4) +2] + inputImage[pixel + (w * 4) +2] + inputImage[pixel - 4 + (w * 4) +2] + inputImage[pixel - 4 +2]) / 6;
	}

else if (threadID > (w * (h - 1)) && threadID < (w * h) - 1) // Bottom Row
	{
		
		r = (inputImage[pixel] + inputImage[pixel - 4] + inputImage[pixel - (w * 4) - 4] + inputImage[pixel - (w * 4)] + inputImage[pixel - (w * 4) + 4]  + inputImage[pixel + 4]) / 6;
		g = (inputImage[pixel + 1] + inputImage[pixel - 4 + 1] + inputImage[pixel - (w * 4) - 4 + 1] + inputImage[pixel - (w * 4) + 1] + inputImage[pixel - (w * 4) + 4 + 1]  + inputImage[pixel + 4 + 1]) / 6;
		b = (inputImage[pixel + 2] + inputImage[pixel - 4 + 2] + inputImage[pixel - (w * 4) - 4 + 2] + inputImage[pixel - (w * 4) + 2] + inputImage[pixel - (w * 4) + 4 + 2]  + inputImage[pixel + 4 + 2]) / 6;
	}

else if (threadID % w == 0) // Left Row
	{
		r = (inputImage[pixel] + inputImage[pixel - (w * 4)] + inputImage[pixel - (w * 4) + 4] + inputImage[pixel + 4] + inputImage[pixel + (w * 4) + 4] + inputImage[pixel + (w * 4)]) / 6;
		g = (inputImage[pixel + 1] + inputImage[pixel - (w * 4) + 1] + inputImage[pixel - (w * 4) + 4 + 1] + inputImage[pixel + 4 + 1] + inputImage[pixel + (w * 4) + 4 + 1] + inputImage[pixel + (w * 4) + 1]) / 6;
		b = (inputImage[pixel + 2] + inputImage[pixel - (w * 4) + 2] + inputImage[pixel - (w * 4) + 4 + 2] + inputImage[pixel + 4 + 2] + inputImage[pixel + (w * 4) + 4 + 2] + inputImage[pixel + (w * 4) + 2]) / 6;
	}

else if (threadID % w == w-1) // Right Side
	{
	
		r = (inputImage[pixel] + inputImage[pixel + (w * 4)] + inputImage[pixel + (w * 4) - 4] + inputImage[pixel - 4] + inputImage[pixel - (w * 4) - 4] + inputImage[pixel - (w * 4)]) / 6;
		g = (inputImage[pixel + 1] + inputImage[pixel + (w * 4) + 1] + inputImage[pixel + (w * 4) - 4 + 1] + inputImage[pixel - 4 + 1] + inputImage[pixel - (w * 4) - 4 + 1] + inputImage[pixel - (w * 4) + 1]) / 6;
		b = (inputImage[pixel + 2] + inputImage[pixel + (w * 4) + 2] + inputImage[pixel + (w * 4) - 4] + 2 + inputImage[pixel - 4 + 2] + inputImage[pixel - (w * 4) - 4 + 2] + inputImage[pixel - (w * 4) + 2]) / 6;
	}
	
else
	{
	
		r = (inputImage[pixel] + inputImage[pixel - 4] + inputImage[pixel - (w * 4) - 4] + inputImage[pixel - (w * 4)] + inputImage[pixel - (w * 4) + 4] + inputImage[pixel + 4] + inputImage[pixel + (w * 4) + 4] + inputImage[pixel + (w * 4)] + inputImage[pixel + (w * 4) - 4]) / 9;
		g = (inputImage[pixel + 1] + inputImage[pixel - 4 + 1] + inputImage[pixel - (w * 4) - 4 + 1] + inputImage[pixel - (w * 4) + 1] + inputImage[pixel - (w * 4) + 4 + 1] + inputImage[pixel + 4 + 1] + inputImage[pixel + (w * 4) + 4 + 1] + inputImage[pixel + (w * 4) + 1] + inputImage[pixel + (w * 4) - 4 + 1]) / 9;
		b = (inputImage[pixel + 2] + inputImage[pixel - 4 + 2] + inputImage[pixel - (w * 4) - 4 + 2] + inputImage[pixel - (w * 4) + 2] + inputImage[pixel - (w * 4) + 4 + 2] + inputImage[pixel + 4 + 2] + inputImage[pixel + (w * 4) + 4 + 2] + inputImage[pixel + (w * 4) + 2] + inputImage[pixel + (w * 4) - 4 + 2]) / 9;
		
	}
	
outputImage[pixel] = r;
outputImage[pixel+1] = g;
outputImage[pixel+2] = b;
outputImage[pixel+3] = a;


}

int main(int argc, char ** argv){

	//decode
	//process
	//encode

	unsigned int errorDecode;
	unsigned char* blurredimage; //hold image values
	unsigned int width, height;

	char * filename = argv[1];
	char *newFilename = argv[2];
	
	errorDecode = lodepng_decode32_file(&blurredimage, &width, &height, filename);
	
	if(errorDecode){
		printf("error %u: %s\n", errorDecode, lodepng_error_text(errorDecode));
	}
	
	int arraySize = width*height*4;
	int memorySize = arraySize * sizeof(unsigned char);
	
	unsigned char* gpuInput;
	unsigned char* gpuOutput;
	
	unsigned int* gpuWidth;
	unsigned int* gpuHeight;
	
	unsigned int* cpuWidth = &width;
	unsigned int* cpuHeight = &height;
	
	unsigned char cpuOutput[arraySize];
	
	hipMalloc( (void**) &gpuInput, memorySize);
	hipMalloc( (void**) &gpuOutput, memorySize);
	
	hipMalloc( (void**) &gpuWidth, sizeof(int));
	hipMalloc( (void**) &gpuHeight, sizeof(int));
	
	hipMemcpy(gpuInput, blurredimage, memorySize, hipMemcpyHostToDevice);
	
	hipMemcpy(gpuWidth, cpuWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuHeight, cpuHeight, sizeof(int), hipMemcpyHostToDevice);

	Blureffect<<< dim3(height,1,1), dim3(width,1,1) >>>(gpuInput, gpuOutput, gpuWidth, gpuHeight);
	hipDeviceSynchronize();
	
	hipMemcpy(cpuOutput, gpuOutput, memorySize, hipMemcpyDeviceToHost);
	
	lodepng_encode32_file(newFilename, cpuOutput, width, height);


	return 0;
}













